// A serial (CPU-based) matrix multiplication program


#include <hip/hip_runtime.h>
#include "stdio.h"

#define SIZE 512

// Matrix multiply function
void multiply(float * A, float * B, float * C) 
{
    
    int i;
    for (i=0;i<SIZE*SIZE;i++) 
        C[i]=0.00; // initialize C to be on the safe side
    
    int row,col,row_len=SIZE,col_len=SIZE;
    for (row=0; row<row_len; row++) {
        
        for (col=0;col < col_len;col++) {
            
            for (i=0;i<SIZE;i++)
                C[row*row_len+col] += A[row*row_len+i]*B[col+i*row_len];
            
        }
        
    }
    
}

int main(int argc, char ** argv) 
{
    // create matrices and initialize A and B to arbitray values
    float A[SIZE*SIZE],B[SIZE*SIZE],C[SIZE*SIZE];
    int i;
    for (i=0;i<SIZE*SIZE;i++) {
        A[i] = (float)i;
        B[i] = (float)SIZE*(float)SIZE-(float)i-1.0;
    }
    
    multiply(A,B,C); // perform matrix multiply

    return 0;
}
