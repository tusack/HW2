// A shared memory matrix multiplication program


#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdlib.h"

// Keep the SIZE evenly divisible by TILE_WIDTH 
#define SIZE 512
#define TILE_WIDTH 16

// kernels that are called by another kernel use the __device__ identifier
__device__ float * GetSubMatrix(float * large_matrix, int row, int col)
{
    // this returns the address of the first element in a sub-matrix
    // when using this sub-matrix you have to jump ahead SIZE elements per row
    float * subMatrix = &large_matrix[ row * SIZE * TILE_WIDTH + col * TILE_WIDTH  ];
    return subMatrix;
}

// matrix multiplication kernel definition
__global__ void MatrixMulKernel(float * A,float * B,float * C)
{

    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    // Here we call another kernel from within a kernel. 
    // Note that we do not need to pass any grid or block
    // information between the <<< >>> symbols.
    float * Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes on element of Csub
    // by accumulating results into Csub
    float Cvalue = 0.0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Iterate through n sub_matrices, where n depends on the size of the tile and matrix
    int sub_matrices_per_row = gridDim.x;

    int i;
    for ( i = 0; i < sub_matrices_per_row; i++ )
    {
        // Get sub-matrices
        // Block threads will work on these sub-matrices
        float * Asub = GetSubMatrix(A, blockRow, i);
        float * Bsub = GetSubMatrix(B, i, blockCol);
        
        // Shared memory used to store Asub and Bsub
        __shared__ float As[TILE_WIDTH][TILE_WIDTH];
        __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

        // Load Asub and Bsub from device memory into shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = Asub[ row*SIZE + col ];
        Bs[row][col] = Bsub[ row*SIZE + col ];

        // Sync threads to make sure sub-matrices are completely loaded into shared memory
        // Remember, this only syncs threads within a block
        __syncthreads();

        int j;
        // Multiply Asub and Bsub together, using fast shared memory for data access
        for ( j = 0; j < TILE_WIDTH ; j++ )
            Cvalue += As[row][j] * Bs[j][col]; 

        // Sync threads to ensure the preceding computation is done before loading
        // two new sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    Csub[ row*SIZE + col ] = Cvalue;
    
}

int main(int argc, char ** argv) 
{

   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
    
   hipEventRecord(start);

   float h_A[SIZE*SIZE],h_B[SIZE*SIZE],h_C[SIZE*SIZE];
   float * d_A, * d_B, * d_C;

   // initialize host matrices with arbitrary data
   int i;
   for (i=0;i<SIZE*SIZE;i++) {
      h_A[i] = (float)i;
      h_B[i] = (float)SIZE * (float)SIZE - (float)i - 1.00;
      h_C[i] = 0.0;
   }

   // allocate space on device
   size_t size = SIZE*SIZE*sizeof(float);
   hipMalloc(&d_A,size);
   hipMalloc(&d_B,size);
   hipMalloc(&d_C,size);

   //copy data to device
   hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
   hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);
   hipMemcpy(d_C,h_C,size,hipMemcpyHostToDevice);

   dim3 threadsPerBlock(TILE_WIDTH,TILE_WIDTH); // 2d block
   int blocks = ( SIZE + TILE_WIDTH - 1 ) / TILE_WIDTH;
   dim3 blocksPerGrid(blocks,blocks); // 2d grid

   // invoke the kernel here
   MatrixMulKernel<<< blocksPerGrid, threadsPerBlock >>>(d_A,d_B,d_C);
   
   // copy results back to host
   hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   float milliseconds = 0;
   hipEventElapsedTime(&milliseconds, start, stop);
   printf("kernel time (ms) : %7.5f\n", milliseconds);

   hipEventDestroy(start);
   hipEventDestroy(stop);

   // Free up device memory
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   return 0;
}
