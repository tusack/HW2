// A tiled matrix multiplication program


#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdlib.h"

#define SIZE 512
#define TILE_WIDTH 16

// kernel definition
__global__ void MatrixMulKernel(float * A,float * B,float * C,int len)
{

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // if SIZE is not evenly divisible by TILE_WIDTH then
    // there will be excess threads that do nothing and should
    // not do anything to prevent out-of-bounds errors
    if ( row >= SIZE || col >= SIZE ) return;

    float sum=0.0;
    int i;
    for (i = 0;i < len;i++) {
        sum += A[ row*len + i ] * B[ i*len + col ];
    }

    C[ row*len + col ] = sum;

}

int main(int argc, char ** argv) 
{
   float h_A[SIZE*SIZE],h_B[SIZE*SIZE],h_C[SIZE*SIZE];
   float * d_A, * d_B, * d_C;

   // initialize host matrices with arbitrary data
   int i;
   for (i=0;i<SIZE*SIZE;i++) {
      h_A[i] = (float)i;
      h_B[i] = (float)SIZE * (float)SIZE - (float)i - 1.00;
      h_C[i] = 0.0;
   }

   // allocate space on device
   size_t size = SIZE*SIZE*sizeof(float);
   hipMalloc(&d_A,size);
   hipMalloc(&d_B,size);
   hipMalloc(&d_C,size);

   //copy data to device
   hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
   hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);
   hipMemcpy(d_C,h_C,size,hipMemcpyHostToDevice);

   dim3 threadsPerBlock(TILE_WIDTH,TILE_WIDTH); // 2d block
   int blocks = ( SIZE + TILE_WIDTH - 1) / TILE_WIDTH;
   dim3 blocksPerGrid(blocks,blocks); // 2d grid

   // invoke the kernel here
   MatrixMulKernel<<< blocksPerGrid, threadsPerBlock >>>(d_A,d_B,d_C,SIZE);

   // copy results back to host
   hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);

   // Free up device memory
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   return 0;
}
